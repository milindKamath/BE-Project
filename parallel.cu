#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <fstream>
#include <sstream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdlib.h>
#include <bitset>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/unique.h>
#include <thrust/execution_policy.h>
#include <thrust/remove.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/find.h>
#include <>
#include <hip/hip_runtime.h>
#include "helper_timer.h"
 
#define MAX_CONF 100 
#define MIN_CONF 0 

#define MAX_SUPP 100 
#define MIN_SUPP 0 

#define WIDTH 55
#define MAX 500


using namespace std;	
using namespace thrust;


__global__ void RearrangeDataPatterns(int *d_array, int *d_new ,int *len,int *item, int size, int count){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if(idx < count){
		//printf("My ID:\t%d\n", idx);
		int k = 0;
		int j = 0;
		while(k < size){
			for(int item_index = idx*WIDTH; item_index < idx*WIDTH+len[idx]; item_index++){
				if(d_array[item_index] == item[k]){
					d_new[idx*WIDTH+j] = item[k];
					j++; 
				}
			}
			k++;
		}
			//if (idx == 0){
			//int i = 0;
			//for (int index_col =idx*WIDTH; index_col <(idx+1)*WIDTH; index_col++){
			//	printf("[%d][%d]  %d..",idx, i++, d_new[56]);
			//	if(idx == 0)
			//		printf("\n");
			//}
			//}
		len[idx] = j ;
					 
	}
}


struct FileProps{
		fstream fp;
		long num_of_transactions ;

		FileProps(){
			num_of_transactions = 0 ;
		}

		//void fileProp(); 
}f_prop;

struct CmdArgs_Type {
		double confidence ;
		double support ;
		char* filename ;
		int conf_val;
		int supp_val; 

		CmdArgs_Type()
		{
			confidence = 80 ;
			support = 80 ;
		}
		//void clac_val();
		
}c_args;

struct compare_support{
	double supp;
	compare_support(double _supp) : supp(_supp) {};
	__host__ __device__
	bool operator()(int& x) const {
		return x < supp;
	}
};

struct ParentRefNode{
	int parentID;
	bool* bitRelation;	
	struct ParentRefNode *ptr;
	int count ;
	
	ParentRefNode(){parentID =0;
	ptr =NULL;};
	
	ParentRefNode(int size){
		this-> bitRelation = new bool [size];
		memset(this->bitRelation, 0, sizeof(bool) *size);
		this-> ptr = NULL;
		this-> count = 0;
	}
	
	void setParentID(int item){
		this->parentID = item;
	}
	
	void setBit(int index){
		this->bitRelation[index] =1;
		this-count++;
	}
};

struct GraphNode{
	int itemID;
	struct ParentRefNode **ParentList;
		
	void init (int item, int size){
		itemID = item;
		this -> ParentList = new ParentRefNode*[size+1];
		for (int index =0; index < size+1; index++)
			ParentList[index] = NULL; 
	}
	
	void init1 (int size){}
};

struct PatternElements{
	int element;
	int frequency;
	struct PatternElements *next;

	PatternElements(){element = 0;}

	PatternElements(int ele){
		element = ele;
		frequency = 1;
		next = NULL;
	}
};


struct CPpattern{
	int itemID;
	struct PatternElements *elements;
	struct CPpattern *next;

	CPpattern(int item){
		itemID = item;
		elements = NULL;
		next = NULL;
	}
};


struct String{
	string str;
	String *next;

	String(){
		str = "DEFAULT";
		next = NULL;
	}
};

void generateCPpattern(CPpattern*, GraphNode**, int, int);
void printCP(CPpattern*);

bool processCmdArgs(int argc, char* argv[]){
	
	if (argc < 2) {
		cerr << "USAGE: " << argv[0] << " FilePath\n" ;
		return false;
	}
	c_args.filename = argv[1] ;
	
	CONF :
	if (argc > 2){
	 c_args.confidence = atof(argv[2]);
	}
	if( c_args.confidence < MIN_CONF || c_args.confidence >= MAX_CONF ){
		cout << "Please enter a value between 1-100" ;
		goto CONF ;	
	}
	SUPP :
	if (argc > 3){
		c_args.support = atof(argv[3]) ;
	}
	if( c_args.support < MIN_CONF || c_args.support >= MAX_CONF ){
		cout << "Please enter a value between 1-100" ;
		goto SUPP ;
	}
		
	return true;
}


void fileProp(){			
	f_prop.fp.open(c_args.filename, ios::out | ios::in);
	if(f_prop.fp == NULL ){
		cout << "File Error : Please check the file path your entered.\n";
	}

	string line;
	while (getline(f_prop.fp,line))
		if (!line.empty())
			++ f_prop.num_of_transactions;
}
		
void calc_val(){
	c_args.conf_val=f_prop.num_of_transactions*(c_args.confidence/100);
	c_args.supp_val=f_prop.num_of_transactions*(c_args.support/100);
}


void rearrangeDataPatterns(int *trans_arr, int* item, int size, int* trans_length)
{
	int *d_array, *d_new, *d_len;
	
	hipMalloc((void **)&d_array, f_prop.num_of_transactions* WIDTH* sizeof(int));
	hipMemcpy(d_array, trans_arr, f_prop.num_of_transactions* WIDTH* sizeof(int), hipMemcpyHostToDevice);
	
	hipMalloc((void **)&d_new, f_prop.num_of_transactions* WIDTH* sizeof(int));
	hipMemset(d_new, 0, f_prop.num_of_transactions* WIDTH* sizeof(int));
	
	hipMalloc((void **)&d_len, f_prop.num_of_transactions*sizeof(int));
	hipMemcpy(d_len, trans_length, f_prop.num_of_transactions*sizeof(int), hipMemcpyHostToDevice);
	
	dim3 GridDim =dim3 (ceil((f_prop.num_of_transactions/1024.0)),1,1);
	dim3 BlockDim = dim3 (1024,1,1);	
	RearrangeDataPatterns<<<GridDim, BlockDim>>>(d_array, d_new, d_len, item, size, f_prop.num_of_transactions);
	hipDeviceSynchronize();
	hipMemcpy(trans_arr, d_new, f_prop.num_of_transactions* WIDTH* sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(trans_length, d_len, f_prop.num_of_transactions*sizeof(int), hipMemcpyDeviceToHost);
	
	/*for(int i=0;i<f_prop.num_of_transactions*WIDTH;i++){
		cout<<trans_arr[i]<<"\t";	
	}*/	
}


float timeForModule1=0.0;
float timeForModule2=0.0;
float timeForModule3=0.0;
float totaltimetaken=0.0;

int main (int argc, char* argv[]){
 
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		fprintf(stderr, "error: no devices supporting CUDA.\n");
		exit(EXIT_FAILURE);
	}
    	int dev = 0;
    	hipSetDevice(dev);

    	hipDeviceProp_t dev_prop;
    	if (hipGetDeviceProperties(&dev_prop, dev) == 0){
    		printf("******************************GPU DETECTED*****************************\n");
    		printf("GPU Device Number                                    : %d\n", deviceCount);
			printf("GPU Device Name                                      : %s\n", dev_prop.name);
			printf("GPU Device Compute Capability                        : %d.%d\n", dev_prop.major, dev_prop.minor);
			printf("GPU Device Clock Rate                                : %d kHz\n", dev_prop.clockRate);
			printf("\n");
     	}
     	
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
     
	cout << "\n!!!!!!~~~~~~ GENERATING GFHTable ~~~~~~!!!!!!\n";
	printf("\n");
	
    
	if(!processCmdArgs(argc, argv))
		return 1;
	fileProp();
	
	calc_val();
	cout << "Total number of transactions = \t" << f_prop.num_of_transactions << endl;
	cout << "Minimum Support =\t" << c_args.supp_val << "\t= "<< c_args.support << "%\n";
	int *trans_arr = new int [f_prop.num_of_transactions*WIDTH];
	int *trans_length = new int[f_prop.num_of_transactions];
	
	fstream fp(c_args.filename, ios::out | ios::in);
	if(fp == NULL ){
		cout << "File Error : Please check the file path your entered.\n";
	}	
	string line;

	int itemize;
	int index= 0 ;
	int count = 0 ;
	while (getline(fp,line)){
		if (!line.empty()){
				int indexy = 0 ;
				index = count * WIDTH;
				istringstream stream(line);				
				while(stream >> itemize ){

					if (itemize != 0){
						trans_arr[index] = itemize;
						index++;
						indexy++;
					}	
				}
			trans_length[count++] = indexy ;
			
		}
	}
	device_vector<int>trans_vect(f_prop.num_of_transactions*55);
	copy_n(trans_arr, f_prop.num_of_transactions*WIDTH, trans_vect.begin());

   	sort(trans_vect.begin(), trans_vect.end());
	
	device_vector<int>item_arr(MAX);
	device_vector<int>freq_arrinit(f_prop.num_of_transactions*WIDTH,1);
	
	reduce_by_key(trans_vect.begin(), trans_vect.end(), make_constant_iterator<long int>(1), item_arr.begin(), freq_arrinit.begin());
	freq_arrinit.resize(MAX);
	sort_by_key(freq_arrinit.begin()+1,freq_arrinit.end(),item_arr.begin()+1,thrust::greater<int>());
	
	device_vector<int>::iterator iter;
	iter=find_if(freq_arrinit.begin(),freq_arrinit.end(),compare_support(c_args.supp_val));
	item_arr.resize(thrust::distance(freq_arrinit.begin(), iter));
	freq_arrinit.resize(thrust::distance(freq_arrinit.begin(), iter));
	 
	
	device_vector<int>item(thrust::distance(freq_arrinit.begin(), iter)-1);
	device_vector<int>freq(thrust::distance(freq_arrinit.begin(), iter)-1);
	
	thrust::copy(item_arr.begin()+1, item_arr.end(), item.begin());
	thrust::copy(freq_arrinit.begin()+1, freq_arrinit.end(), freq.begin());
	
	item_arr.clear();
	item_arr.shrink_to_fit();
	freq_arrinit.clear();
	freq_arrinit.shrink_to_fit();
	trans_vect.clear();
	trans_vect.shrink_to_fit();
	
	int *raw_ptr = raw_pointer_cast(item.data());
	int *raw_ptr1 = raw_pointer_cast(trans_vect.data());
	for(int index = 0; index<item.size(); index++)
		cout << "Item[" << index << "]" << item[index] << "\t->" << freq[index]<< endl;
		
	cout << "\n!!!!!!~~~~~~ GENERATED GFHTable ~~~~~~!!!!!!\n";
	
	
		
	rearrangeDataPatterns(trans_arr, raw_ptr, item.size(), trans_length);
	sdkStopTimer(&timer);
    timeForModule1 = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    
    cout<<"Time taken for GFHTable creation: "<<timeForModule1<<"ms\t==>"<<timeForModule1/1000<<" seconds\n";
    totaltimetaken=timeForModule1;
	
	host_vector<int>item1(item);
	
	
	//end of module 1
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------
	
	
	StopWatchInterface *timer1 = NULL;
    sdkCreateTimer(&timer1);
    sdkStartTimer(&timer1);	
		
	cout << "\n!!!!!!~~~~~~ START OF GRAPH CREATION ~~~~~~!!!!!!\n";	
	//for (int indext = 0; indext < f_prop.num_of_transactions; indext++)	  			
	//	cout << trans_length[indext]<< endl;
    
	GraphNode **graph = new GraphNode *[item1.size()];
	
	for(int index=0; index <item1.size(); index++){
		GraphNode *temp = new GraphNode();
		temp->init(item1[index], index);
		graph[index] = new GraphNode();
		graph[index] = temp;
	}
	int i;
	//for(int index=0; index <item1.size(); index++)
	//	cout << graph[index]-> itemID<< endl;
	ParentRefNode *temp;
	for (int indext = 0; indext < f_prop.num_of_transactions; indext++){
		GraphNode *parent = NULL;
		for (int index = 0; index < trans_length[indext]; index++){
			for (i =0; i < item1.size(); i++){
				if (item1[i] == trans_arr[indext*WIDTH+index]){
					if (parent == NULL){
						break;
					}
					else{
						if(graph[i]->ParentList[0] == NULL){
							temp = new ParentRefNode(f_prop.num_of_transactions);
							temp->setParentID(parent->itemID);
							temp->setBit(indext);
							graph[i]->ParentList[0] = temp;
//							cout << "Parent:\t" << graph[i]->ParentList[0]->parentID<<endl;
							break;		
						}
						
						else{
							for (int indexy =0; indexy < i; indexy++){ 
								if(graph[i]->ParentList[indexy] != NULL){
									if (graph[i]->ParentList[indexy]->parentID == parent->itemID){
										graph[i]->ParentList[indexy]->setBit(indext);
							//			cout << "COUNT:\t"<<graph[i]->ParentList[indexy]->count ;
										break;
									}
								}
								
								
								if(graph[i]->ParentList[indexy] == NULL){
									temp = new ParentRefNode(f_prop.num_of_transactions);
									temp->setParentID(parent->itemID);
									temp->setBit(indext);
									graph[i]->ParentList[indexy] = temp;
								//	cout << "Parent:\t" << graph[i]->ParentList[indexy]->parentID<<endl;
									break;
								}
								
							}
							break;
						}
							
					}
				}
			}
			parent = graph[i];
		}
	}
	
	cout<<endl<<item.size() -1 <<" nodes created\n";
/*	
	for(int index=0; index <item1.size(); index++){
	int ind = 0;
		while(graph[index]->ParentList[ind] != NULL){
			cout << "\n\n....." << graph[index]->itemID;
			cout << "\t"<< graph[index]->ParentList[ind]->parentID;
			cout << "\t"<< graph[index]->ParentList[ind]->count;
			ind++;
		}
	}
*/
		cout << "\n!!!!!!~~~~~~ GRAPH CREATED ~~~~~~!!!!!!\n";
		
		sdkStopTimer(&timer1);
    	timeForModule2 = sdkGetTimerValue(&timer1);
    	sdkDeleteTimer(&timer1);
    
   	    cout<<"Time taken for Graph creation: "<<timeForModule2<<"ms\t==>"<<timeForModule2/1000<<" seconds\n";
    	totaltimetaken=totaltimetaken+timeForModule2;
    
    
    	//end of module 2
    	//----------------------------------------------------------------------------------------------------------------------------------------------------------
    
	
		
		StopWatchInterface *timer2 = NULL;
    	sdkCreateTimer(&timer2);
    	sdkStartTimer(&timer2);
	
		CPpattern *head = NULL;
		CPpattern *curr= new CPpattern(0);
		CPpattern *nextnode;
		for (int i = 0; i < item1.size(); i++){
			if(head == NULL){
				head = new CPpattern(item1[i]);
				curr = head;
			}
			else{
				nextnode = new CPpattern(item1[i]);
				curr->next = nextnode;
				curr = nextnode;
			}
			generateCPpattern(curr,graph,i,item1.size());
		}	
		cout << "\n\n!!!!!!~~~~~~ CP PATTERNS GENERATED ~~~~~~!!!!!!\n";
		CPpattern *iterator = head;
		while(iterator != NULL){
			cout << iterator-> itemID;
			printCP(iterator);
			iterator = iterator-> next;
			cout << endl;
		}
		
	sdkStopTimer(&timer2);
    timeForModule3 = sdkGetTimerValue(&timer2);
    sdkDeleteTimer(&timer2);
    
   	cout<<"Time taken for CP Pattern creation: "<<timeForModule3<<"ms\t==>"<<timeForModule3/1000<<" seconds\n";
    totaltimetaken=totaltimetaken+timeForModule3;
    		
    cout<<"#################################################\n";
	cout<<"Time taken for Program execution: "<<totaltimetaken<<"ms\t==>"<<totaltimetaken/1000<<" seconds\n";
	}


void fillcp(CPpattern *curr, GraphNode **graph, int transid, int newindex, int size){
	PatternElements *newelement = new PatternElements(graph[newindex]->itemID);
	
	PatternElements *iter = curr->elements;
	if(curr->elements == NULL){
		curr-> elements = newelement;
		goto added;
	}
	while(iter-> next != NULL){
		iter = iter-> next;
	}
	iter-> next = newelement;

	// element added

	added: 
	
	if (graph[newindex]-> ParentList[0] == NULL)
		return;
	int nindex = 0;
	for (int index = 0; graph[newindex]-> ParentList[index] != NULL; index++){
		if(graph[newindex]-> ParentList[index]-> bitRelation[transid]){
			for (int j = 0; j < size; j++){
				if(graph[j]-> itemID==graph[newindex]-> ParentList[index]-> parentID){
					nindex = j;
					break;
				}
			}
			fillcp(curr, graph, transid, nindex, size);
		}
	}
	return;

}

void checkcp(CPpattern *curr, GraphNode **graph, int transid, int newindex, int size){
//	cout<<"Supestcheck"<<transid<<endl;
	PatternElements *iter = curr->elements;
	if(curr->elements == NULL){
		goto next;
	}	
	while(iter != NULL){
		if(iter-> element == graph[newindex]-> itemID){
			iter-> frequency++;
			goto next;
		}
		iter = iter-> next;
	}
	next:
//	int parentpath =0;
	if (graph[newindex]-> ParentList[0] == NULL)
		return;
	int nindex = 0;
	for (int index = 0; graph[newindex]-> ParentList[index] != NULL; index++){
		if(graph[newindex]-> ParentList[index]-> bitRelation[transid]){
			for (int id = 0; id < size; id++){
				if(graph[id]-> itemID == graph[newindex]-> ParentList[index]->  parentID){
					nindex = id;
					break;
				}
			}
			checkcp(curr,graph,transid,nindex, size);
		}
	}
}


void pruneCP(CPpattern* curr, int numPaths){
	PatternElements *current, *previous;

	current = curr-> elements;
	previous = NULL;

	while(current != NULL){
		if (current-> frequency != numPaths){
			if(previous == NULL){
				curr-> elements = current-> next;
				current-> next = NULL;
				current = curr-> elements;
			}
			else{
				previous-> next = current-> next;
				current = current-> next;
			}
		}
		else{
			previous = current;
			current = current-> next;
		}
	}
}



void generateCPpattern(CPpattern *curr, GraphNode **graph, int i, int size){
	if(graph[i]-> ParentList[0]==NULL){
		return;
	}

	int multipath = 0;
	int newindex = 0;
	bool firsttime = true;
/*
	for (int index = 0; graph[i]->ParentList[index]!=NULL; index++){
		cout << graph[i]-> itemID;
	}
	cout << endl<<endl;
*/
	for (int index = 0; graph[i]-> ParentList[index] != NULL; index++){
		for (int j = 0; j < size; j++){
			if(graph[j]-> itemID == graph[i]-> ParentList[index]-> parentID){
				newindex = j;
				break;
			}
		}
		int pathcounts = graph[i]-> ParentList[index]-> count;
		multipath += pathcounts;
		int transid = 0;
		int count = 0;
		while(pathcounts != 0){
			while(count < f_prop.num_of_transactions){
				if(graph[i]-> ParentList[index]-> bitRelation[count]){
					transid = count;
					break;
				}
				count++;
			}
			if(firsttime){
				fillcp(curr, graph, transid, newindex, size);
				firsttime = false;
			}
			else{
				checkcp(curr, graph, transid, newindex, size);
			}	
			count++;
			pathcounts--;
		}
	}

	if(multipath > 1){
		pruneCP(curr, multipath);
	}
}

int generateCombinations(string *inputSet, int inIndex, string prefixSet, string** outputSet, int outIndex, int count){
	string tempSet = "";
	int localIndex = inIndex;
	int index = 0;
	while(localIndex < count){
		tempSet = prefixSet + inputSet[localIndex] + " " ;
		outputSet[outIndex][index] = tempSet;
		outIndex = generateCombinations(inputSet, localIndex+1, tempSet, outputSet, outIndex+1, count); 
		localIndex++;
	}
	return outIndex;
} 

void getCombinations(string inputSet, int count, CPpattern *iter){
	if(inputSet.length() == 0)
		return;
	string* inputSetSend = new string [count];
	int inIndex = 0;
	for ( int index = 0; index < count ;){
		if(inputSet[inIndex] != ' ')	
			inputSetSend[index] += inputSet[inIndex];
		else 
			index++; 
		inIndex++;
	}
			
	string **outputSet;
	int len =  pow(2, count) - 1;
	outputSet = new string* [len];
	for ( int index = 0; index < len ; index++ )
		outputSet[index] = new string;
	generateCombinations (inputSetSend, 0, "", outputSet, 0, count);
	for ( int index = 0; index < len ; index++ ){
		cout << "{" << iter-> itemID << " " <<  outputSet[index][0]  << "}"<< endl;
	}
}

void printCP(CPpattern *iter){
	cout << "-> ";
	cout << "{" << iter-> itemID << "}";
	string patternSet;
	int count = 0;
	PatternElements *indexIter = iter-> elements;
	while(indexIter != NULL){
		stringstream ss;
		ss << indexIter-> element;
		patternSet += ss.str();
		patternSet += " ";
		count ++;
		indexIter = indexIter-> next;
	}
//	cout << patternSet <<endl ;
	
	getCombinations(patternSet, count, iter);
}
